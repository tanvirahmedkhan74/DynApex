#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "array_sum.h"

__global__ void sumArrayOnDevice(float* A, float* B, float* C, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

extern "C" void sumArrayOnGPU(float* A, float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * sizeof(float));
    hipMalloc((void**)&d_B, N * sizeof(float));
    hipMalloc((void**)&d_C, N * sizeof(float));
    
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);
    
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    sumArrayOnDevice<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
    
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
